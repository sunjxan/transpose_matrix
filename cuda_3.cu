#include "hip/hip_runtime.h"
#include "common.hpp"

// 使用__ldg加载

__global__ void kernel(const real (*A)[N], real (*B)[M])
{
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (iy < N && ix < M) {
        B[iy][ix] = __ldg(&A[ix][iy]);
    }
}

void transpose_matrix(const real *A, real *B)
{
    const real (*nA)[N] = reinterpret_cast<decltype(nA)>(A);
    real (*nB)[M] = reinterpret_cast<decltype(nB)>(B);

    dim3 block_size(32, 32);
    // N是列对应y，M是行对应x
    dim3 grid_size(DIVUP(M, block_size.x), DIVUP(N, block_size.y));
    kernel<<<grid_size, block_size>>>(nA, nB);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}