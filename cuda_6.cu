#include "hip/hip_runtime.h"
#include "common.hpp"

// 使用填充共享内存，避免bank冲突

__global__ void kernel(const real (*A)[N], real (*B)[M], size_t sld)
{
    unsigned ty = threadIdx.y, bdy = blockDim.y, iy = blockIdx.y * bdy + ty;
    unsigned tx = threadIdx.x, bdx = blockDim.x, ix = blockIdx.x * bdx + tx;

    extern __shared__ real s_a[];

    if (iy < M && ix < N) {
        s_a[ty * sld + tx] = A[iy][ix];
    } 
    __syncthreads();

    unsigned niy = ix - tx + ty;
    unsigned nix = iy - ty + tx;

    if (niy < N && nix < M) {
        B[niy][nix] = s_a[tx * sld + ty];
    }
}

void transpose_matrix(const real *A, real *B)
{
    const real (*nA)[N] = reinterpret_cast<decltype(nA)>(A);
    real (*nB)[M] = reinterpret_cast<decltype(nB)>(B);

    // block_size应是正方形
    dim3 block_size(32, 32);
    // N是列对应x，M是行对应y
    dim3 grid_size(DIVUP(N, block_size.x), DIVUP(M, block_size.y));
    // 共享内存列数添加pad，sld表示shared memory leading dimension
    size_t pad = 1, sld = block_size.x + pad;
    kernel<<<grid_size, block_size, block_size.y * sld * real_size>>>(nA, nB, sld);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}